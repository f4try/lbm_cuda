#include "hip/hip_runtime.h"
//#include <hip/hip_runtime.h>
#include <>
#include "cpu_bitmap.h"
#include "cu_complex_cuda.h"
#define DIM 500
__device__ int julia(int x, int y) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);
	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);
	int i = 0;
	for (i = 0; i < 200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}
	return 1;
}
__global__ void kernel(unsigned char* ptr) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	int juliaValue = julia(x, y);
	ptr[offset * 4 + 0] = 255 * juliaValue;
	ptr[offset * 4 + 1] = 0;
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
}
int main() {
	CPUBitmap bitmap(DIM, DIM);
	unsigned char* dev_bitmap;
	hipMalloc((void**)&dev_bitmap, bitmap.image_size());
	dim3 grid(DIM, DIM);
	kernel<<<grid, 1>>>(dev_bitmap);
	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size()
		, hipMemcpyDeviceToHost);
	bitmap.display_and_exit();
	hipFree(dev_bitmap);
}