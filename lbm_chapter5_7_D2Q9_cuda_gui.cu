#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "cpu_anim.h"
#define m 256
#define n 256
#define alpha 0.25f
#define omega (1.0 / (3.0 * alpha + 0.5))
#define twall 1.0f
#define nstep 200
struct DataBlock {
	unsigned char* output_bitmap;
	CPUAnimBitmap* bitmap;
	float* dev_rho;
	float* dev_f;
	float* dev_f_old;
	float* dev_w;
	float* dev_e;
	float frames;
};

__global__ void collision(float* f_old, float* f, float* w, float* rho) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int k = blockIdx.z;

	f_old[y * m * 9 + x * 9 + k] = (1 - omega) * f[y * m * 9 + x * 9 + k] + omega * w[k] * rho[y * m + x];
}
__global__ void stream(float* f_old, float* f, float* e) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int k = blockIdx.z;
	int xp = x - e[k * 2 + 0];
	int yp = y - e[k * 2 + 1];
	if (xp >= 0 && xp < m && yp >= 0 && yp < n) {
		f[y * m * 9 + x * 9 + k] = f_old[yp * m * 9 + xp * 9 + k];
	}

}
__global__ void boundary_tb(float* f) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	f[0 * m * 9 + x * 9 + 2] = f[1 * m * 9 + x * 9 + 2];
	f[0 * m * 9 + x * 9 + 5] = f[1 * m * 9 + x * 9 + 5];
	f[0 * m * 9 + x * 9 + 6] = f[1 * m * 9 + x * 9 + 6];

	f[(n - 1) * m * 9 + x * 9 + 7] = -f[(n - 1) * m * 9 + x * 9 + 5];
	f[(n - 1) * m * 9 + x * 9 + 4] = -f[(n - 1) * m * 9 + x * 9 + 2];
	f[(n - 1) * m * 9 + x * 9 + 8] = -f[(n - 1) * m * 9 + x * 9 + 6];
}
__global__ void boundary_lr(float* f, float* w) {
	int y = threadIdx.x + blockIdx.x * blockDim.x;
	f[y * m * 9 + 0 * 9 + 1] = w[1] * twall + w[3] * twall - f[y * m * 9 + 0 * 9 + 3];
	f[y * m * 9 + 0 * 9 + 5] = w[5] * twall + w[7] * twall - f[y * m * 9 + 0 * 9 + 7];
	f[y * m * 9 + 0 * 9 + 8] = w[8] * twall + w[6] * twall - f[y * m * 9 + 0 * 9 + 6];

	f[y * m * 9 + (m - 1) * 9 + 3] = -f[y * m * 9 + (m - 1) * 9 + 1];
	f[y * m * 9 + (m - 1) * 9 + 7] = -f[y * m * 9 + (m - 1) * 9 + 5];
	f[y * m * 9 + (m - 1) * 9 + 6] = -f[y * m * 9 + (m - 1) * 9 + 8];
}
__global__ void update(float* rho, float* f) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	rho[y * m + x] = f[y * m * 9 + x * 9 + 0] + f[y * m * 9 + x * 9 + 1] + f[y * m * 9 + x * 9 + 2] + f[y * m * 9 + x * 9 + 3] + f[y * m * 9 + x * 9 + 4] + f[y * m * 9 + x * 9 + 5] + f[y * m * 9 + x * 9 + 6] + f[y * m * 9 + x * 9 + 7] + f[y * m * 9 + x * 9 + 8];
}

__global__ void rho_to_bitmap(unsigned char* bitmap, float* rho) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * m;
	if (rho[offset]>=0.5 && rho[offset]<=1) {
		bitmap[offset * 4 + 0] = 255 * 2 * (rho[offset]-0.5);
		bitmap[offset * 4 + 1] = 255 * 2 * (1.0 - rho[offset]);
		bitmap[offset * 4 + 2] = 0;
		bitmap[offset * 4 + 3] = 255;
	}
	else if (rho[offset] < 0.5 && rho[offset] >=0) {
		bitmap[offset * 4 + 0] = 0;
		bitmap[offset * 4 + 1] = 255 * 2 * rho[offset];
		bitmap[offset * 4 + 2] = 255 * 2 * (0.5 - rho[offset]);
		bitmap[offset * 4 + 3] = 255;
	}
	/*if (y <= rho[(n-1)/2*m+x] * (m - 50)) {
		bitmap[offset * 4 + 0] = 255;
		bitmap[offset * 4 + 1] = 0;
		bitmap[offset * 4 + 2] = 0;
		bitmap[offset * 4 + 3] = 255;
	}*/
}

void anim_gpu(DataBlock* d, int ticks) {
	CPUAnimBitmap* bitmap = d->bitmap;
	dim3 grid2d(m / 16, n / 16);
	dim3 grid3d(m / 16, n / 16, 9);
	dim3 threads(16, 16);
	for (int k = 0; k < nstep; k++) {
		collision << <grid3d, threads >> > (d->dev_f_old, d->dev_f, d->dev_w, d->dev_rho);
		hipDeviceSynchronize();
		stream << <grid3d, threads >> > (d->dev_f_old, d->dev_f, d->dev_e);
		hipDeviceSynchronize();
		boundary_tb << <n / 16, 16 >> > (d->dev_f);
		hipDeviceSynchronize();
		boundary_lr << <m / 16, 16 >> > (d->dev_f, d->dev_w);
		hipDeviceSynchronize();
		update << <grid2d, threads >> > (d->dev_rho, d->dev_f);
		hipDeviceSynchronize();
	}
	rho_to_bitmap << <grid2d, threads >> > (d->output_bitmap, d->dev_rho);
	hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);
	++d->frames;
}

void solve(DataBlock* d) {
	dim3 grid2d(m / 16, n / 16);
	dim3 grid3d(m / 16, n / 16, 9);
	dim3 threads(16, 16);
	for (int k = 0; k < nstep; k++) {
		collision << <grid3d, threads >> > (d->dev_f_old, d->dev_f, d->dev_w, d->dev_rho);
		hipDeviceSynchronize();
		stream << <grid3d, threads >> > (d->dev_f_old, d->dev_f, d->dev_e);
		hipDeviceSynchronize();
		boundary_tb << <n / 16, 16 >> > (d->dev_f);
		hipDeviceSynchronize();
		boundary_lr << <m / 16, 16 >> > (d->dev_f, d->dev_w);
		hipDeviceSynchronize();
		update << <grid2d, threads >> > (d->dev_rho, d->dev_f);
		hipDeviceSynchronize();
	}
}
void print(float* rho) {
	/*for (int y = 0; y < m; y++) {
		for (int x = 0; x < 2; x++) {
			std::cout << rho[y * m + x] << ",";
		}
		std::cout << std::endl;
	}*/
	for (int x = 0; x < m; x++) {
		std::cout <<  rho[(n - 1) / 2 * m + x] << std::endl;
	}
}
void anim_exit(DataBlock* d) {
	hipFree(d->dev_rho);
	hipFree(d->dev_f);
	hipFree(d->dev_f_old);
	hipFree(d->dev_e);
	hipFree(d->dev_w);
}
int main() {
	float w[9] = { 4. / 9., 1. / 9., 1. / 9., 1. / 9., 1. / 9.
			, 1. / 36., 1. / 36., 1. / 36., 1. / 36. };
	float e[9 * 2] = { 0, 0, 1, 0, 0, 1, -1, 0, 0, -1, 1, 1,
		-1, 1, -1, -1, 1, -1 };
	float* rho, * f, * f_old;
	DataBlock data;
	CPUAnimBitmap bitmap((int)m, (int)n, (void*)&data);
	data.bitmap = &bitmap;
	data.frames = 0;
	rho = new float[m * n];
	f = new float[m * n * 9];
	f_old = new float[m * n * 9];
	for (int x = 0; x < m; x++) {
		for (int y = 0; y < n; y++) {
			rho[x * n + y] = 0;
			for (int k = 0; k < 9; k++) {
				f[x * n * 9 + y * 9 + k] = 0;
				f_old[y * m * 9 + x * 9 + k] = 0;
			}
		}
	}
	hipMalloc((void**)&data.output_bitmap, bitmap.image_size());
	hipMalloc((void**)&data.dev_rho, sizeof(float) * m * n);
	hipMalloc((void**)&data.dev_f, sizeof(float) * m * n * 9);
	hipMalloc((void**)&data.dev_f_old, sizeof(float) * m * n * 9);
	hipMalloc((void**)&data.dev_w, sizeof(float) * 9);
	hipMalloc((void**)&data.dev_e, sizeof(float) * 9 * 2);

	hipMemcpy(data.dev_rho, rho, sizeof(float) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(data.dev_f, f, sizeof(float) * m * n * 9, hipMemcpyHostToDevice);
	hipMemcpy(data.dev_f_old, f_old, sizeof(float) * m * n * 9, hipMemcpyHostToDevice);
	hipMemcpy(data.dev_w, w, sizeof(float) * 9, hipMemcpyHostToDevice);
	hipMemcpy(data.dev_e, e, sizeof(float) * 9 * 2, hipMemcpyHostToDevice);

	data.bitmap->anim_and_exit((void(*)(void*, int))anim_gpu, (void(*)(void*))anim_exit);
	//solve(&data);
	hipMemcpy(rho, data.dev_rho, sizeof(float) * m * n, hipMemcpyDeviceToHost);
	print(rho);
	
	delete[] f;
	delete[] f_old;
	delete[] rho;
}