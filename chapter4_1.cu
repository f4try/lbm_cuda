
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10
void add(int* a, int* b, int* c) {
	int tid = 0;
	while (tid < N) {
		c[tid] = a[tid] + b[tid];
		tid += 1;
	}
}
int main() {
	int a[N], b[N], c[N];
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	add(a, b, c);
	for (int i = 0; i < N; i++) {
		printf("%d+%d=%d\n", a[i], b[i], c[i]);
	}
	return 0;
}