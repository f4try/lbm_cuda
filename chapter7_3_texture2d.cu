#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
#include "cpu_anim.h"

#define DIM 1024
#define SPEED 0.25f
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f

texture<float, 2> texConstSrc;
texture<float, 2> texIn;
texture<float, 2> texOut;
struct DataBlock {
	unsigned char* output_bitmap;
	float* dev_inSrc;
	float* dev_outSrc;
	float* dev_constSrc;
	CPUAnimBitmap* bitmap;
	hipEvent_t start, stop;
	float totalTime;
	float frames;
};
__device__ unsigned char value(float n1, float n2, int hue) {
	if (hue > 360) hue -= 360;
	else if (hue < 0) hue += 360;
	if (hue < 60)
		return (unsigned char)(255 * (n1 + (n2 - n1) * hue / 60));
	if (hue < 180)
		return (unsigned char)(255 * n2);
	if (hue < 240)
		return (unsigned char)(255 * (n1 + (n2 - n1) * (240 - hue) / 60));

}
__global__ void float_to_color(unsigned char* optr, const float* outSrc) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float l = outSrc[offset];
	float s = 1;
	int h = (180 + (int)(360.0f * outSrc[offset])) % 360;
	float m1, m2;
	if (l <= 0.5) {
		m2 = l * (1 + s);
	}
	m1 = 2 * l - m2;
	optr[offset * 4 + 0] = value(m1, m2, h + 120);
	optr[offset * 4 + 1] = value(m1, m2, h);
	optr[offset * 4 + 2] = value(m1, m2, h - 120);
	optr[offset * 4 + 3] = 255;
}

__global__ void copy_const_kernel(float* iptr) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float c = tex2D(texConstSrc, x,y);
	if (c != 0) {
		iptr[offset] = c;
	}
}

__global__ void blend_kernel(float* dst, bool dstOut) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float t, l, c, r, b;
	if (dstOut) {
		t = tex2D(texIn, x, y - 1);
		l = tex2D(texIn, x - 1, y);
		c = tex2D(texIn, x, y);
		r = tex2D(texIn, x + 1, y);
		b = tex2D(texIn, x, y + 1);
	}
	else {
		t = tex2D(texOut, x, y - 1);
		l = tex2D(texOut, x - 1, y);
		c = tex2D(texOut, x, y);
		r = tex2D(texOut, x + 1, y);
		b = tex2D(texOut, x, y + 1);
	}
	dst[offset] = c + SPEED * (t + b + l + r - c * 4);
}

void anim_gpu(DataBlock* d, int ticks) {
	hipEventRecord(d->start, 0);
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	CPUAnimBitmap* bitmap = d->bitmap;
	volatile bool dstOut = true;
	for (int i = 0; i < 90; i++) {
		float* in, * out;
		if (dstOut) {
			in = d->dev_inSrc;
			out = d->dev_outSrc;
		}
		else {
			out = d->dev_inSrc;
			in = d->dev_outSrc;
		}
		copy_const_kernel << <blocks, threads >> > (in);
		blend_kernel << <blocks, threads >> > (out, dstOut);
		dstOut = !dstOut;
	}
	float_to_color << <blocks, threads >> > (d->output_bitmap, d->dev_inSrc);
	hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);
	hipEventRecord(d->stop, 0);
	hipEventSynchronize(d->stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, d->start, d->stop);
	d->totalTime += elapsedTime;
	++d->frames;
	printf("Average time per frame: %3.1f ms\n", d->totalTime / d->frames);
}
void anim_exit(DataBlock* d) {
	hipUnbindTexture(texIn);
	hipUnbindTexture(texOut);
	hipUnbindTexture(texConstSrc);
	hipFree(d->dev_inSrc);
	hipFree(d->dev_outSrc);
	hipFree(d->dev_constSrc);
	hipEventDestroy(d->start);
	hipEventDestroy(d->stop);
}

int main() {
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;
	hipEventCreate(&data.start);
	hipEventCreate(&data.stop);
	hipMalloc((void**)&data.output_bitmap, bitmap.image_size());
	hipMalloc((void**)&data.dev_inSrc, bitmap.image_size());
	hipMalloc((void**)&data.dev_outSrc, bitmap.image_size());
	hipMalloc((void**)&data.dev_constSrc, bitmap.image_size());

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	hipBindTexture2D(NULL, texConstSrc, data.dev_constSrc, desc, DIM, DIM, sizeof(float) * DIM);
	hipBindTexture2D(NULL, texIn, data.dev_inSrc, desc, DIM, DIM, sizeof(float) * DIM);
	hipBindTexture2D(NULL, texOut, data.dev_outSrc, desc, DIM, DIM, sizeof(float) * DIM);

	float* temp = (float*)malloc(bitmap.image_size());
	for (int i = 0; i < DIM * DIM; i++) {
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x > 300) && (x < 600) && (y > 310) && (y < 601)) {
			temp[i] = MAX_TEMP;
		}
	}
	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;
	for (int y = 800; y < 900; y++) {
		for (int x = 400; x < 500; x++) {
			temp[x + y * DIM] = MIN_TEMP;
		}
	}
	hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
	for (int y = 800; y < DIM; y++) {
		for (int x = 0; x < 200; x++) {
			temp[x + y * DIM] = MAX_TEMP;
		}
	}
	hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
	bitmap.anim_and_exit((void(*)(void*, int))anim_gpu, (void(*)(void*))anim_exit);

}
