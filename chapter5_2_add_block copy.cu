#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#define N (33*1024)

__global__ void add(int* a, int* b, int* c) {
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	while (tid < N) {
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
	
}
int main() {
	int a[N], b[N], c[N];
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * i;
	}
	int* dev_a, * dev_b, * dev_c;
	hipMalloc((void**)&dev_a, sizeof(int) * N);
	hipMalloc((void**)&dev_b, sizeof(int) * N);
	hipMalloc((void**)&dev_c, sizeof(int) * N);
	hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
	add << <256, 256 >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}