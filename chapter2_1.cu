#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

__global__ void VecAdd(float* A,float* B,float* C){
    int i = threadIdx.x
    C[i] = A[i] + B[i]
}
int main(){
    VecAdd<<<1,N>>>(A,B,C);
}