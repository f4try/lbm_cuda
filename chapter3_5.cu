#include <iostream>
#include <hip/hip_runtime.h>

int main(void) {
	hipDeviceProp_t prop;
	int dev;
	hipGetDevice(&dev);
	printf("ID of current CUDA device: %d\n", dev);
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 8;
	prop.minor = 6;
	hipChooseDevice(&dev, &prop);
	printf("ID of CUDA device closest to revision 8.6: %d\n", dev);
	hipSetDevice(dev);
}