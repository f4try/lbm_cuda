#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <iostream>
#include <math.h>
#define PI 3.1415927
void framebuffer_size_callback(GLFWwindow* window, int width, int height) {
	glViewport(0, 0, width, height);
}
void processInput(GLFWwindow* window) {
	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS) {
		glfwSetWindowShouldClose(window, true);
	}
}
const float vertices[] = {
	 0.5f, 0.5f,0.0f,
	 0.5f,-0.5f,0.0f,
	-0.5f,-0.5f,0.0f,
	-0.5f, 0.5f,0.0f,
};
const unsigned int indices[] = {
	0,1,3,
	1,2,3
};
const char* vertexShaderSource = "#version 330 core\n"
	"layout (location =0) in vec3 aPos;\n"
	"void main()\n"
	"{\n"
	"	gl_Position = vec4(aPos.x,aPos.y,aPos.z,1.0);\n"
	"}\0";
const char* fragmentShaderSource = "#version 330 core\n"
	"out vec4 FragColor;\n"
	"void main(){\n"
	"FragColor=vec4(1.0f,0.5f,0.2f,1.0f);"
	"}\0";
int main() {
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	GLFWwindow* window = glfwCreateWindow(800, 600, "xxzz", NULL, NULL);
	if (window == NULL) {
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
		std::cout << "Failed to initialize GLAD" << std::endl;
		return -1;
	}
	glViewport(0, 0, 800, 600);
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
	unsigned int VBO;
	glGenBuffers(1, &VBO);
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
	unsigned int vertexShader;
	vertexShader = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
	glCompileShader(vertexShader);
	int success;
	char infoLog[512];
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success) {
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
	}
	unsigned int fragmentShader;
	fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
	glCompileShader(fragmentShader);
	unsigned int shaderProgram;
	shaderProgram = glCreateProgram();
	glAttachShader(shaderProgram, vertexShader);
	glAttachShader(shaderProgram, fragmentShader);
	glLinkProgram(shaderProgram);
	glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
	if (!success) {
		glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
		std::cout << "ERROR::program\n" << infoLog << std::endl;
	}
	glUseProgram(shaderProgram);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	unsigned int VAO;
	glGenVertexArrays(1, &VAO);
	glBindVertexArray(VAO);
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	unsigned int EBO;
	glGenBuffers(1, &EBO);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);
	//glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	//glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	float t = 0;
	float vertices_dynamic[] = {0.5*cos(t),0.5 * sin(t),0,
								0.5 * cos(t+2*PI/3),0.5 * sin(t + 2 * PI / 3),0,
								0.5 * cos(t + 4 * PI / 3),0.5 * sin(t + 4 * PI / 3),0};
	while (!glfwWindowShouldClose(window)) {
		processInput(window);
		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);
		glUseProgram(shaderProgram);
		glBindVertexArray(VAO);
		for (int i = 0; i < 3;i++) {
			vertices_dynamic[i * 3] = 0.5 * cos(t+ i * 2 * PI/3);
			vertices_dynamic[i * 3+1] = 0.5 * sin(t + i * 2 * PI / 3);
		}
		t+=0.02;
		glBufferData(GL_ARRAY_BUFFER, sizeof(vertices_dynamic), vertices_dynamic, GL_STATIC_DRAW);
		glDrawArrays(GL_TRIANGLES, 0, 3);
		//glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
		//glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
		glfwSwapBuffers(window);
		glfwPollEvents();
	}
	glDeleteShader(vertexShader);
	glDeleteShader(fragmentShader);
	glfwTerminate();
	return 0;
}