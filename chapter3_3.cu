#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add(int a, int b, int* c) {
	*c = a + b;
}
int main() {
	int c;
	int* dev_c;
	hipMalloc((int**)&dev_c, sizeof(int));
	add << <1, 1 >> > (2, 7, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	std::cout << c << std::endl;
	hipFree(dev_c);
	return 0;
}